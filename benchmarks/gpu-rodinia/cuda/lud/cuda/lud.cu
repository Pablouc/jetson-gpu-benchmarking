/*
 * =====================================================================================
 *
 *       Filename:  lud.cu
 *
 *    Description:  The main wrapper for the suite
 *
 *        Version:  1.0
 *        Created:  10/22/2009 08:40:34 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Liang Wang (lw2aw), lw2aw@virginia.edu
 *        Company:  CS@UVa
 *
 * =====================================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>
#include <stdlib.h>
#include <assert.h>
#include "lud.h"
#include "common.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


#ifdef TIMING
#include "timing.h"
#endif

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif



static int do_verify = 0;

static struct option long_options[] = {
  /* name, has_arg, flag, val */
  {"input", 1, NULL, 'i'},
  {"size", 1, NULL, 's'},
  {"verify", 0, NULL, 'v'},
  {0,0,0,0}
};

extern void
lud_cuda(float *d_m, int matrix_dim, hipStream_t stream);

#ifdef TIMING
struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;
#endif

void lud_main(const char* input_file, int matrix_dim, int do_verify,  hipStream_t stream) {

	printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
	func_ret_t ret;
	float *m, *d_m, *mm;
        stopwatch sw;

	if (input_file) {

		printf("Reading matrix from file %s\n", input_file);
		ret = create_matrix_from_file(&m, input_file, &matrix_dim);

		if (ret != RET_SUCCESS) {

			m = NULL;
			fprintf(stderr, "error create matrix from file %s\n", input_file);
			exit(EXIT_FAILURE);

		}

	} else if (matrix_dim) {

		printf("Creating matrix internally size=%d\n", matrix_dim);
		ret = create_matrix(&m, matrix_dim);
		
		if (ret != RET_SUCCESS) {

			m = NULL;
			fprintf(stderr, "error create matrix internally size=%d\n", matrix_dim);
			exit(EXIT_FAILURE);

		}

	} else {

		printf("Invalid input parameters!\n");
		exit(EXIT_FAILURE);
	
	}

	if (do_verify) {

		printf("Before LUD\n");
		// print_matrix(m, matrix_dim);
		matrix_duplicate(m, &mm, matrix_dim);

	}

	hipMalloc((void**)&d_m, matrix_dim * matrix_dim * sizeof(float));
	hipMemcpy(d_m, m, matrix_dim * matrix_dim * sizeof(float), hipMemcpyHostToDevice);



       	stopwatch_start(&sw); // Start the stopwatch

#ifdef TIMING
	gettimeofday(&tv_kernel_start, NULL);
#endif


	lud_cuda(d_m, matrix_dim, stream);

#ifdef TIMING
	gettimeofday(&tv_kernel_end, NULL);
	tvsub(&tv_kernel_end, &tv_kernel_start, &tv);
	kernel_time += tv.tv_sec * 1000.0 + (float)tv.tv_usec / 1000.0;
#endif


	hipMemcpy(m, d_m, matrix_dim * matrix_dim * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_m);

	stopwatch_stop(&sw); // Stop the stopwatch

	if (do_verify) {

		printf("After LUD\n");
		// print_matrix(m, matrix_dim);
		printf(">>>Verify<<<<\n");
		lud_verify(mm, m, matrix_dim);
		free(mm);

	}


	free(m);

#ifdef TIMING
	printf("Exec: %f\n", kernel_time);
#endif

	printf("Time consumed(ms): %lf\n", 1000 * get_interval_by_sec(&sw));
}

