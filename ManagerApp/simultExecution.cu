#include <stdlib.h>
#include <stdio.h>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include "structs.h"
#include "simultExecution.h"

using namespace std;
#include "/home/carpab00/Desktop/Pablo/jetson-gpu-benchmarking/benchmarks/gpu-rodinia/cuda/bfs/bfs.h"
#include "/home/carpab00/Desktop/Pablo/jetson-gpu-benchmarking/benchmarks/gpu-rodinia/cuda/lud/cuda/lud.h"

// Workloads Paths
char* bfsWorkloadPath = "/home/carpab00/Desktop/Pablo/jetson-gpu-benchmarking/benchmarks/gpu-rodinia/data/bfs/"; //graph65536.txt





int simultExecution(Configuration& config) {

// Print or use the configuration variables here

	     	cout << "Simult: " << config.simult << endl;
			cout << "Blocks: " << config.blocks << endl;
				cout << "Threads: " << config.threads << endl;
					cout << "Frequency: " << config.frequency << endl;

   char* concatenatedPath = new char[200];


   //Verify how many apps to execute
   int appsCount = 0;
  
   /* for (int i = 0; i < ; ++i) {
	   // Check if the app at index i is non-empty
	   if (config.apps[i][0] != '\0') {
		   appsCount++;
	   }
	   cout <<" app iteration"<< i <<endl;
   }*/


   // Allocate an array of hipStream_t
    hipStream_t* streams = new hipStream_t[config.appsNum];
   

 // Initialize each stream
     for (int i = 0; i < config.appsNum ; i++){	  
	     hipStreamCreate(&streams[i]); // Create a new stream and store its handle in the array
     }


  // Events for measuring time
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   for( int i = 0 ; i < config.appsNum; i++){

	   hipEventRecord(start);
           
	   if(strcmp(config.apps[i], "bfs") == 0 ){
		   strcpy(concatenatedPath, bfsWorkloadPath);
		   strcat(concatenatedPath, config.workloads[i]);
		   cout << "ConcatenatedPath: " << concatenatedPath << endl;
		   BFSGraph(concatenatedPath, config.blocks, config.threads, streams[i]);

	   } 

	   const char* input_file = NULL;  // Set your input file name
	   int matrix_dim = 768;  // Set your matrix dimension
	   int do_verify = 1;
	   char makeCommand[200]; // adjust the size according to your needs
	       
	   int rdWgSizeValue = 64; // The value you want to set for RD_WG_SIZE_0_0

	    int cleanResult = system("make -C /home/carpab00/Desktop/Pablo/jetson-gpu-benchmarking/benchmarks/gpu-rodinia/cuda/lud/cuda/ clean");

	           
	       // Construct the make command with the specific variable value
	           snprintf(makeCommand, sizeof(makeCommand), "make -C /home/carpab00/Desktop/Pablo/jetson-gpu-benchmarking/benchmarks/gpu-rodinia/cuda/lud/cuda/ BLOCK_SIZE=%d", rdWgSizeValue);

		       // Execute the make command with system()
		       int result = system(makeCommand);

	   	       
	   // Check the result of the system() call	   
       	   if (result == 0) {
		               
		   // The command executed successfully	  
       		   cout << "Make command executed successfully.\n";
	   } else {
					           
		   // There was an error executing the command
		   cerr << "Error executing make command.\n";

	   }


           lud_main(input_file, matrix_dim, do_verify, streams[i]);
	   hipEventRecord(stop);
	   hipEventSynchronize(stop);

   }

		
   float milliseconds = 0;

   hipEventElapsedTime(&milliseconds, start, stop);

   std::cout << "Total execution time:  " << milliseconds << " ms\n";
   return 0;
}
