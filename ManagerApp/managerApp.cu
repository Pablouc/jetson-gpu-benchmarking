#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include "parseLine.h"
using namespace std;

  

int managerApp(const char* configFile) {

	ifstream file(configFile);
	if (!file.is_open()) {
	    cerr << "Error: Cannot open file." << endl;
	    return 1;
	}


	Configuration config;
	memset(&config, 0, sizeof(Configuration)); // Initialize the struct

	char line[256];
    	int lineCount = 0;
	while (file.getline(line, sizeof(line))) {
		++lineCount;
		if (parseLine(line, config)) {
			cerr << "Error at line " << lineCount << endl;
			file.close();
			return 1;

		}

	}
	
	// Print or use the configuration variables here

     	cout << "Simult: " << config.simult << endl;
	cout << "Blocks: " << config.blocks << endl;
	cout << "Threads: " << config.threads << endl;
	cout << "Frequency: " << config.frequency << endl;
	cout << "Apps: ";

	for (int i = 0; i < 8 && config.apps[i] != nullptr; ++i) {
		cout << config.apps[i] << " ";

	}

	cout << endl;

	cout << "Workloads: ";

	for (int i = 0; i < 8 && config.workloads[i] != nullptr; ++i) {

		cout << config.workloads[i] << " ";

	}

	cout << endl;


	
	// Free allocated memory for strings

	for (int i = 0; i < 8; ++i) {

		if (config.apps[i] != nullptr) {

			free(config.apps[i]);

		}

		if (config.workloads[i] != nullptr) {

			free(config.workloads[i]);

		}

	}


	
	file.close();

	return 0;
}
	


int main() {
    const char* inputFileName = "exec_config.txt";
    managerApp(inputFileName);
    return 0;
}

