
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <termio.h>
#include <unistd.h>
#include <time.h>
using namespace std;
int main(){
 ifstream gpuPowerFile;
 int totalCount=0;
 int sumofPower=0;
 char input=0;
 time_t start=time(NULL);
 while(true){
   string str;
   int power=0;
   int voltage=0;
   int val;
   gpuPowerFile.open("/sys/bus/i2c/drivers/ina3221x/1-0040/iio:device0/in_power0_input");


   gpuPowerFile>>str;
   val=atoi(str.c_str());
   cout<<"GPU Power:"<<val<<"mW"<<endl;
   power+=val;
   str.clear();
  
   gpuPowerFile.close();
   
   cout<<"Total Power:"<<power<<"mW"<<endl;
   sumofPower+=power;
   totalCount++;
   cout<<"AveragePower: "<<sumofPower/totalCount<<endl;
   time_t end=time(NULL);
   cout<<"Time: "<<(double)(end-start)<<endl;
   sleep(1);
 }
}